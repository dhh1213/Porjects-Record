#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"
using namespace std;
#define BLOCK_WIDTH 16
__constant__ float deviceKernel[4096];

#define nStreams 50
hipStream_t stream[nStreams];

__global__ void conv_forward_kernel_4(float *__restrict y, const float *__restrict x, const int B, const int M, const int C, const int H, const int W, const int K, const int offset)
{
    /*
    Modify this function to implement the forward pass described in Chapter 16.
    We have added an additional dimension to the tensors to support an entire mini-batch
    The goal here is to be correct AND fast.

    Function paramter definitions:
    y - output
    x - input
    k - kernel
    B - batch_size (number of images in x)
    M - number of output feature maps
    C - number of input feature maps
    H - input height dimension
    W - input width dimension
    K - kernel height and width (K x K)
    */

    const int H_out = H - K + 1;
    const int W_out = W - K + 1;
    //(void)H_out; // silence declared but never referenced warning. remove this line when you start working
    //(void)W_out; // silence declared but never referenced warning. remove this line when you start working

    // We have some nice #defs for you below to simplify indexing. Feel free to use them, or create your own.
    // An example use of these macros:
    // float a = y4d(0,0,0,0)
    // y4d(0,0,0,0) = a

#define y4d(i3, i2, i1, i0) y[(i3) * (M * H_out * W_out) + (i2) * (H_out * W_out) + (i1) * (W_out) + i0]
#define x4d(i3, i2, i1, i0) x[(i3) * (4 * H * W) + (i2) * (H * W) + (i1) * (W) + i0]
#define k4d(i3, i2, i1, i0) deviceKernel[(i3) * (4 * K * K) + (i2) * (K * K) + (i1) * (K) + i0]

    // Insert your GPU convolution kernel code here
	int W_grid = ceil(W_out/float(BLOCK_WIDTH));
	
	int n = blockIdx.x+offset;
	int m = blockIdx.y;
	int h_id = (blockIdx.z/W_grid)*BLOCK_WIDTH + threadIdx.y;
	int w_id = (blockIdx.z%W_grid)*BLOCK_WIDTH + threadIdx.x;

	if( w_id < W_out && h_id < H_out){
		// sum over all input feature maps
			// KxK filter
			// C = 0
			y4d(n, m, h_id, w_id) = 
			x4d(n, 0, h_id + 0, w_id + 0) * k4d(m, 0, 0, 0)
			 + x4d(n, 0, h_id + 0, w_id + 1) * k4d(m, 0, 0, 1)
			 + x4d(n, 0, h_id + 0, w_id + 2) * k4d(m, 0, 0, 2)
			 + x4d(n, 0, h_id + 0, w_id + 3) * k4d(m, 0, 0, 3)
			 + x4d(n, 0, h_id + 0, w_id + 4) * k4d(m, 0, 0, 4)
			 + x4d(n, 0, h_id + 0, w_id + 5) * k4d(m, 0, 0, 5)
			 + x4d(n, 0, h_id + 0, w_id + 6) * k4d(m, 0, 0, 6)
			 + x4d(n, 0, h_id + 1, w_id + 0) * k4d(m, 0, 1, 0)
			 + x4d(n, 0, h_id + 1, w_id + 1) * k4d(m, 0, 1, 1)
			 + x4d(n, 0, h_id + 1, w_id + 2) * k4d(m, 0, 1, 2)
			 + x4d(n, 0, h_id + 1, w_id + 3) * k4d(m, 0, 1, 3)
			 + x4d(n, 0, h_id + 1, w_id + 4) * k4d(m, 0, 1, 4)
			 + x4d(n, 0, h_id + 1, w_id + 5) * k4d(m, 0, 1, 5)
			 + x4d(n, 0, h_id + 1, w_id + 6) * k4d(m, 0, 1, 6)
			 + x4d(n, 0, h_id + 2, w_id + 0) * k4d(m, 0, 2, 0)
			 + x4d(n, 0, h_id + 2, w_id + 1) * k4d(m, 0, 2, 1)
			 + x4d(n, 0, h_id + 2, w_id + 2) * k4d(m, 0, 2, 2)
			 + x4d(n, 0, h_id + 2, w_id + 3) * k4d(m, 0, 2, 3)
			 + x4d(n, 0, h_id + 2, w_id + 4) * k4d(m, 0, 2, 4)
			 + x4d(n, 0, h_id + 2, w_id + 5) * k4d(m, 0, 2, 5)
			 + x4d(n, 0, h_id + 2, w_id + 6) * k4d(m, 0, 2, 6)
			 + x4d(n, 0, h_id + 3, w_id + 0) * k4d(m, 0, 3, 0)
			 + x4d(n, 0, h_id + 3, w_id + 1) * k4d(m, 0, 3, 1)
			 + x4d(n, 0, h_id + 3, w_id + 2) * k4d(m, 0, 3, 2)
			 + x4d(n, 0, h_id + 3, w_id + 3) * k4d(m, 0, 3, 3)
			 + x4d(n, 0, h_id + 3, w_id + 4) * k4d(m, 0, 3, 4)
			 + x4d(n, 0, h_id + 3, w_id + 5) * k4d(m, 0, 3, 5)
			 + x4d(n, 0, h_id + 3, w_id + 6) * k4d(m, 0, 3, 6)
			 + x4d(n, 0, h_id + 4, w_id + 0) * k4d(m, 0, 4, 0)
			 + x4d(n, 0, h_id + 4, w_id + 1) * k4d(m, 0, 4, 1)
			 + x4d(n, 0, h_id + 4, w_id + 2) * k4d(m, 0, 4, 2)
			 + x4d(n, 0, h_id + 4, w_id + 3) * k4d(m, 0, 4, 3)
			 + x4d(n, 0, h_id + 4, w_id + 4) * k4d(m, 0, 4, 4)
			 + x4d(n, 0, h_id + 4, w_id + 5) * k4d(m, 0, 4, 5)
			 + x4d(n, 0, h_id + 4, w_id + 6) * k4d(m, 0, 4, 6)
			 + x4d(n, 0, h_id + 5, w_id + 0) * k4d(m, 0, 5, 0)
			 + x4d(n, 0, h_id + 5, w_id + 1) * k4d(m, 0, 5, 1)
			 + x4d(n, 0, h_id + 5, w_id + 2) * k4d(m, 0, 5, 2)
			 + x4d(n, 0, h_id + 5, w_id + 3) * k4d(m, 0, 5, 3)
			 + x4d(n, 0, h_id + 5, w_id + 4) * k4d(m, 0, 5, 4)
			 + x4d(n, 0, h_id + 5, w_id + 5) * k4d(m, 0, 5, 5)
			 + x4d(n, 0, h_id + 5, w_id + 6) * k4d(m, 0, 5, 6)
			 + x4d(n, 0, h_id + 6, w_id + 0) * k4d(m, 0, 6, 0)
			 + x4d(n, 0, h_id + 6, w_id + 1) * k4d(m, 0, 6, 1)
			 + x4d(n, 0, h_id + 6, w_id + 2) * k4d(m, 0, 6, 2)
			 + x4d(n, 0, h_id + 6, w_id + 3) * k4d(m, 0, 6, 3)
			 + x4d(n, 0, h_id + 6, w_id + 4) * k4d(m, 0, 6, 4)
			 + x4d(n, 0, h_id + 6, w_id + 5) * k4d(m, 0, 6, 5)
			 + x4d(n, 0, h_id + 6, w_id + 6) * k4d(m, 0, 6, 6)
			// C = 1
			 + x4d(n, 1, h_id + 0, w_id + 0) * k4d(m, 1, 0, 0)
			 + x4d(n, 1, h_id + 0, w_id + 1) * k4d(m, 1, 0, 1)
			 + x4d(n, 1, h_id + 0, w_id + 2) * k4d(m, 1, 0, 2)
			 + x4d(n, 1, h_id + 0, w_id + 3) * k4d(m, 1, 0, 3)
			 + x4d(n, 1, h_id + 0, w_id + 4) * k4d(m, 1, 0, 4)
			 + x4d(n, 1, h_id + 0, w_id + 5) * k4d(m, 1, 0, 5)
			 + x4d(n, 1, h_id + 0, w_id + 6) * k4d(m, 1, 0, 6)
			 + x4d(n, 1, h_id + 1, w_id + 0) * k4d(m, 1, 1, 0)
			 + x4d(n, 1, h_id + 1, w_id + 1) * k4d(m, 1, 1, 1)
			 + x4d(n, 1, h_id + 1, w_id + 2) * k4d(m, 1, 1, 2)
			 + x4d(n, 1, h_id + 1, w_id + 3) * k4d(m, 1, 1, 3)
			 + x4d(n, 1, h_id + 1, w_id + 4) * k4d(m, 1, 1, 4)
			 + x4d(n, 1, h_id + 1, w_id + 5) * k4d(m, 1, 1, 5)
			 + x4d(n, 1, h_id + 1, w_id + 6) * k4d(m, 1, 1, 6)
			 + x4d(n, 1, h_id + 2, w_id + 0) * k4d(m, 1, 2, 0)
			 + x4d(n, 1, h_id + 2, w_id + 1) * k4d(m, 1, 2, 1)
			 + x4d(n, 1, h_id + 2, w_id + 2) * k4d(m, 1, 2, 2)
			 + x4d(n, 1, h_id + 2, w_id + 3) * k4d(m, 1, 2, 3)
			 + x4d(n, 1, h_id + 2, w_id + 4) * k4d(m, 1, 2, 4)
			 + x4d(n, 1, h_id + 2, w_id + 5) * k4d(m, 1, 2, 5)
			 + x4d(n, 1, h_id + 2, w_id + 6) * k4d(m, 1, 2, 6)
			 + x4d(n, 1, h_id + 3, w_id + 0) * k4d(m, 1, 3, 0)
			 + x4d(n, 1, h_id + 3, w_id + 1) * k4d(m, 1, 3, 1)
			 + x4d(n, 1, h_id + 3, w_id + 2) * k4d(m, 1, 3, 2)
			 + x4d(n, 1, h_id + 3, w_id + 3) * k4d(m, 1, 3, 3)
			 + x4d(n, 1, h_id + 3, w_id + 4) * k4d(m, 1, 3, 4)
			 + x4d(n, 1, h_id + 3, w_id + 5) * k4d(m, 1, 3, 5)
			 + x4d(n, 1, h_id + 3, w_id + 6) * k4d(m, 1, 3, 6)
			 + x4d(n, 1, h_id + 4, w_id + 0) * k4d(m, 1, 4, 0)
			 + x4d(n, 1, h_id + 4, w_id + 1) * k4d(m, 1, 4, 1)
			 + x4d(n, 1, h_id + 4, w_id + 2) * k4d(m, 1, 4, 2)
			 + x4d(n, 1, h_id + 4, w_id + 3) * k4d(m, 1, 4, 3)
			 + x4d(n, 1, h_id + 4, w_id + 4) * k4d(m, 1, 4, 4)
			 + x4d(n, 1, h_id + 4, w_id + 5) * k4d(m, 1, 4, 5)
			 + x4d(n, 1, h_id + 4, w_id + 6) * k4d(m, 1, 4, 6)
			 + x4d(n, 1, h_id + 5, w_id + 0) * k4d(m, 1, 5, 0)
			 + x4d(n, 1, h_id + 5, w_id + 1) * k4d(m, 1, 5, 1)
			 + x4d(n, 1, h_id + 5, w_id + 2) * k4d(m, 1, 5, 2)
			 + x4d(n, 1, h_id + 5, w_id + 3) * k4d(m, 1, 5, 3)
			 + x4d(n, 1, h_id + 5, w_id + 4) * k4d(m, 1, 5, 4)
			 + x4d(n, 1, h_id + 5, w_id + 5) * k4d(m, 1, 5, 5)
			 + x4d(n, 1, h_id + 5, w_id + 6) * k4d(m, 1, 5, 6)
			 + x4d(n, 1, h_id + 6, w_id + 0) * k4d(m, 1, 6, 0)
			 + x4d(n, 1, h_id + 6, w_id + 1) * k4d(m, 1, 6, 1)
			 + x4d(n, 1, h_id + 6, w_id + 2) * k4d(m, 1, 6, 2)
			 + x4d(n, 1, h_id + 6, w_id + 3) * k4d(m, 1, 6, 3)
			 + x4d(n, 1, h_id + 6, w_id + 4) * k4d(m, 1, 6, 4)
			 + x4d(n, 1, h_id + 6, w_id + 5) * k4d(m, 1, 6, 5)
			 + x4d(n, 1, h_id + 6, w_id + 6) * k4d(m, 1, 6, 6)
			// C = 2
			 + x4d(n, 2, h_id + 0, w_id + 0) * k4d(m, 2, 0, 0)
			 + x4d(n, 2, h_id + 0, w_id + 1) * k4d(m, 2, 0, 1)
			 + x4d(n, 2, h_id + 0, w_id + 2) * k4d(m, 2, 0, 2)
			 + x4d(n, 2, h_id + 0, w_id + 3) * k4d(m, 2, 0, 3)
			 + x4d(n, 2, h_id + 0, w_id + 4) * k4d(m, 2, 0, 4)
			 + x4d(n, 2, h_id + 0, w_id + 5) * k4d(m, 2, 0, 5)
			 + x4d(n, 2, h_id + 0, w_id + 6) * k4d(m, 2, 0, 6)
			 + x4d(n, 2, h_id + 1, w_id + 0) * k4d(m, 2, 1, 0)
			 + x4d(n, 2, h_id + 1, w_id + 1) * k4d(m, 2, 1, 1)
			 + x4d(n, 2, h_id + 1, w_id + 2) * k4d(m, 2, 1, 2)
			 + x4d(n, 2, h_id + 1, w_id + 3) * k4d(m, 2, 1, 3)
			 + x4d(n, 2, h_id + 1, w_id + 4) * k4d(m, 2, 1, 4)
			 + x4d(n, 2, h_id + 1, w_id + 5) * k4d(m, 2, 1, 5)
			 + x4d(n, 2, h_id + 1, w_id + 6) * k4d(m, 2, 1, 6)
			 + x4d(n, 2, h_id + 2, w_id + 0) * k4d(m, 2, 2, 0)
			 + x4d(n, 2, h_id + 2, w_id + 1) * k4d(m, 2, 2, 1)
			 + x4d(n, 2, h_id + 2, w_id + 2) * k4d(m, 2, 2, 2)
			 + x4d(n, 2, h_id + 2, w_id + 3) * k4d(m, 2, 2, 3)
			 + x4d(n, 2, h_id + 2, w_id + 4) * k4d(m, 2, 2, 4)
			 + x4d(n, 2, h_id + 2, w_id + 5) * k4d(m, 2, 2, 5)
			 + x4d(n, 2, h_id + 2, w_id + 6) * k4d(m, 2, 2, 6)
			 + x4d(n, 2, h_id + 3, w_id + 0) * k4d(m, 2, 3, 0)
			 + x4d(n, 2, h_id + 3, w_id + 1) * k4d(m, 2, 3, 1)
			 + x4d(n, 2, h_id + 3, w_id + 2) * k4d(m, 2, 3, 2)
			 + x4d(n, 2, h_id + 3, w_id + 3) * k4d(m, 2, 3, 3)
			 + x4d(n, 2, h_id + 3, w_id + 4) * k4d(m, 2, 3, 4)
			 + x4d(n, 2, h_id + 3, w_id + 5) * k4d(m, 2, 3, 5)
			 + x4d(n, 2, h_id + 3, w_id + 6) * k4d(m, 2, 3, 6)
			 + x4d(n, 2, h_id + 4, w_id + 0) * k4d(m, 2, 4, 0)
			 + x4d(n, 2, h_id + 4, w_id + 1) * k4d(m, 2, 4, 1)
			 + x4d(n, 2, h_id + 4, w_id + 2) * k4d(m, 2, 4, 2)
			 + x4d(n, 2, h_id + 4, w_id + 3) * k4d(m, 2, 4, 3)
			 + x4d(n, 2, h_id + 4, w_id + 4) * k4d(m, 2, 4, 4)
			 + x4d(n, 2, h_id + 4, w_id + 5) * k4d(m, 2, 4, 5)
			 + x4d(n, 2, h_id + 4, w_id + 6) * k4d(m, 2, 4, 6)
			 + x4d(n, 2, h_id + 5, w_id + 0) * k4d(m, 2, 5, 0)
			 + x4d(n, 2, h_id + 5, w_id + 1) * k4d(m, 2, 5, 1)
			 + x4d(n, 2, h_id + 5, w_id + 2) * k4d(m, 2, 5, 2)
			 + x4d(n, 2, h_id + 5, w_id + 3) * k4d(m, 2, 5, 3)
			 + x4d(n, 2, h_id + 5, w_id + 4) * k4d(m, 2, 5, 4)
			 + x4d(n, 2, h_id + 5, w_id + 5) * k4d(m, 2, 5, 5)
			 + x4d(n, 2, h_id + 5, w_id + 6) * k4d(m, 2, 5, 6)
			 + x4d(n, 2, h_id + 6, w_id + 0) * k4d(m, 2, 6, 0)
			 + x4d(n, 2, h_id + 6, w_id + 1) * k4d(m, 2, 6, 1)
			 + x4d(n, 2, h_id + 6, w_id + 2) * k4d(m, 2, 6, 2)
			 + x4d(n, 2, h_id + 6, w_id + 3) * k4d(m, 2, 6, 3)
			 + x4d(n, 2, h_id + 6, w_id + 4) * k4d(m, 2, 6, 4)
			 + x4d(n, 2, h_id + 6, w_id + 5) * k4d(m, 2, 6, 5)
			 + x4d(n, 2, h_id + 6, w_id + 6) * k4d(m, 2, 6, 6)
			// C = 3
			 + x4d(n, 3, h_id + 0, w_id + 0) * k4d(m, 3, 0, 0)
			 + x4d(n, 3, h_id + 0, w_id + 1) * k4d(m, 3, 0, 1)
			 + x4d(n, 3, h_id + 0, w_id + 2) * k4d(m, 3, 0, 2)
			 + x4d(n, 3, h_id + 0, w_id + 3) * k4d(m, 3, 0, 3)
			 + x4d(n, 3, h_id + 0, w_id + 4) * k4d(m, 3, 0, 4)
			 + x4d(n, 3, h_id + 0, w_id + 5) * k4d(m, 3, 0, 5)
			 + x4d(n, 3, h_id + 0, w_id + 6) * k4d(m, 3, 0, 6)
			 + x4d(n, 3, h_id + 1, w_id + 0) * k4d(m, 3, 1, 0)
			 + x4d(n, 3, h_id + 1, w_id + 1) * k4d(m, 3, 1, 1)
			 + x4d(n, 3, h_id + 1, w_id + 2) * k4d(m, 3, 1, 2)
			 + x4d(n, 3, h_id + 1, w_id + 3) * k4d(m, 3, 1, 3)
			 + x4d(n, 3, h_id + 1, w_id + 4) * k4d(m, 3, 1, 4)
			 + x4d(n, 3, h_id + 1, w_id + 5) * k4d(m, 3, 1, 5)
			 + x4d(n, 3, h_id + 1, w_id + 6) * k4d(m, 3, 1, 6)
			 + x4d(n, 3, h_id + 2, w_id + 0) * k4d(m, 3, 2, 0)
			 + x4d(n, 3, h_id + 2, w_id + 1) * k4d(m, 3, 2, 1)
			 + x4d(n, 3, h_id + 2, w_id + 2) * k4d(m, 3, 2, 2)
			 + x4d(n, 3, h_id + 2, w_id + 3) * k4d(m, 3, 2, 3)
			 + x4d(n, 3, h_id + 2, w_id + 4) * k4d(m, 3, 2, 4)
			 + x4d(n, 3, h_id + 2, w_id + 5) * k4d(m, 3, 2, 5)
			 + x4d(n, 3, h_id + 2, w_id + 6) * k4d(m, 3, 2, 6)
			 + x4d(n, 3, h_id + 3, w_id + 0) * k4d(m, 3, 3, 0)
			 + x4d(n, 3, h_id + 3, w_id + 1) * k4d(m, 3, 3, 1)
			 + x4d(n, 3, h_id + 3, w_id + 2) * k4d(m, 3, 3, 2)
			 + x4d(n, 3, h_id + 3, w_id + 3) * k4d(m, 3, 3, 3)
			 + x4d(n, 3, h_id + 3, w_id + 4) * k4d(m, 3, 3, 4)
			 + x4d(n, 3, h_id + 3, w_id + 5) * k4d(m, 3, 3, 5)
			 + x4d(n, 3, h_id + 3, w_id + 6) * k4d(m, 3, 3, 6)
			 + x4d(n, 3, h_id + 4, w_id + 0) * k4d(m, 3, 4, 0)
			 + x4d(n, 3, h_id + 4, w_id + 1) * k4d(m, 3, 4, 1)
			 + x4d(n, 3, h_id + 4, w_id + 2) * k4d(m, 3, 4, 2)
			 + x4d(n, 3, h_id + 4, w_id + 3) * k4d(m, 3, 4, 3)
			 + x4d(n, 3, h_id + 4, w_id + 4) * k4d(m, 3, 4, 4)
			 + x4d(n, 3, h_id + 4, w_id + 5) * k4d(m, 3, 4, 5)
			 + x4d(n, 3, h_id + 4, w_id + 6) * k4d(m, 3, 4, 6)
			 + x4d(n, 3, h_id + 5, w_id + 0) * k4d(m, 3, 5, 0)
			 + x4d(n, 3, h_id + 5, w_id + 1) * k4d(m, 3, 5, 1)
			 + x4d(n, 3, h_id + 5, w_id + 2) * k4d(m, 3, 5, 2)
			 + x4d(n, 3, h_id + 5, w_id + 3) * k4d(m, 3, 5, 3)
			 + x4d(n, 3, h_id + 5, w_id + 4) * k4d(m, 3, 5, 4)
			 + x4d(n, 3, h_id + 5, w_id + 5) * k4d(m, 3, 5, 5)
			 + x4d(n, 3, h_id + 5, w_id + 6) * k4d(m, 3, 5, 6)
			 + x4d(n, 3, h_id + 6, w_id + 0) * k4d(m, 3, 6, 0)
			 + x4d(n, 3, h_id + 6, w_id + 1) * k4d(m, 3, 6, 1)
			 + x4d(n, 3, h_id + 6, w_id + 2) * k4d(m, 3, 6, 2)
			 + x4d(n, 3, h_id + 6, w_id + 3) * k4d(m, 3, 6, 3)
			 + x4d(n, 3, h_id + 6, w_id + 4) * k4d(m, 3, 6, 4)
			 + x4d(n, 3, h_id + 6, w_id + 5) * k4d(m, 3, 6, 5)
			 + x4d(n, 3, h_id + 6, w_id + 6) * k4d(m, 3, 6, 6);
	}
#undef y4d
#undef x4d
#undef k4d
}
__global__ void conv_forward_kernel_1(float *__restrict y, const float *__restrict x, const int B, const int M, const int C, const int H, const int W, const int K, const int offset)
{

    const int H_out = H - K + 1;
    const int W_out = W - K + 1;

#define y4d(i3, i2, i1, i0) y[(i3) * (M * H_out * W_out) + (i2) * (H_out * W_out) + (i1) * (W_out) + i0]
#define x4d(i3, i2, i1, i0) x[(i3) * (1 * H * W) + (i1) * (W) + i0]
#define k4d(i3, i2, i1, i0) deviceKernel[(i3) * (1 * K * K) + (i1) * (K) + i0]

    // Insert your GPU convolution kernel code here
	int W_grid = ceil(W_out/float(BLOCK_WIDTH));
	
	int n = blockIdx.x+offset;
	int m = blockIdx.y;
	int h_id = (blockIdx.z/W_grid)*BLOCK_WIDTH + threadIdx.y;
	int w_id = (blockIdx.z%W_grid)*BLOCK_WIDTH + threadIdx.x;

	if( w_id < W_out && h_id < H_out){
		y4d(n, m, h_id, w_id) = 
		 x4d(n, 0, h_id + 0, w_id + 0) * k4d(m, 0, 0, 0)
		 + x4d(n, 0, h_id + 0, w_id + 1) * k4d(m, 0, 0, 1)
		 + x4d(n, 0, h_id + 0, w_id + 2) * k4d(m, 0, 0, 2)
		 + x4d(n, 0, h_id + 0, w_id + 3) * k4d(m, 0, 0, 3)
		 + x4d(n, 0, h_id + 0, w_id + 4) * k4d(m, 0, 0, 4)
		 + x4d(n, 0, h_id + 0, w_id + 5) * k4d(m, 0, 0, 5)
		 + x4d(n, 0, h_id + 0, w_id + 6) * k4d(m, 0, 0, 6)
											  
		 + x4d(n, 0, h_id + 1, w_id + 0) * k4d(m, 0, 1, 0)
		 + x4d(n, 0, h_id + 1, w_id + 1) * k4d(m, 0, 1, 1)
		 + x4d(n, 0, h_id + 1, w_id + 2) * k4d(m, 0, 1, 2)
		 + x4d(n, 0, h_id + 1, w_id + 3) * k4d(m, 0, 1, 3)
		 + x4d(n, 0, h_id + 1, w_id + 4) * k4d(m, 0, 1, 4)
		 + x4d(n, 0, h_id + 1, w_id + 5) * k4d(m, 0, 1, 5)
		 + x4d(n, 0, h_id + 1, w_id + 6) * k4d(m, 0, 1, 6)
											  
		 + x4d(n, 0, h_id + 2, w_id + 0) * k4d(m, 0, 2, 0)
		 + x4d(n, 0, h_id + 2, w_id + 1) * k4d(m, 0, 2, 1)
		 + x4d(n, 0, h_id + 2, w_id + 2) * k4d(m, 0, 2, 2)
		 + x4d(n, 0, h_id + 2, w_id + 3) * k4d(m, 0, 2, 3)
		 + x4d(n, 0, h_id + 2, w_id + 4) * k4d(m, 0, 2, 4)
		 + x4d(n, 0, h_id + 2, w_id + 5) * k4d(m, 0, 2, 5)
		 + x4d(n, 0, h_id + 2, w_id + 6) * k4d(m, 0, 2, 6)
											  
		 + x4d(n, 0, h_id + 3, w_id + 0) * k4d(m, 0, 3, 0)
		 + x4d(n, 0, h_id + 3, w_id + 1) * k4d(m, 0, 3, 1)
		 + x4d(n, 0, h_id + 3, w_id + 2) * k4d(m, 0, 3, 2)
		 + x4d(n, 0, h_id + 3, w_id + 3) * k4d(m, 0, 3, 3)
		 + x4d(n, 0, h_id + 3, w_id + 4) * k4d(m, 0, 3, 4)
		 + x4d(n, 0, h_id + 3, w_id + 5) * k4d(m, 0, 3, 5)
		 + x4d(n, 0, h_id + 3, w_id + 6) * k4d(m, 0, 3, 6)
											  
		 + x4d(n, 0, h_id + 4, w_id + 0) * k4d(m, 0, 4, 0)
		 + x4d(n, 0, h_id + 4, w_id + 1) * k4d(m, 0, 4, 1)
		 + x4d(n, 0, h_id + 4, w_id + 2) * k4d(m, 0, 4, 2)
		 + x4d(n, 0, h_id + 4, w_id + 3) * k4d(m, 0, 4, 3)
		 + x4d(n, 0, h_id + 4, w_id + 4) * k4d(m, 0, 4, 4)
		 + x4d(n, 0, h_id + 4, w_id + 5) * k4d(m, 0, 4, 5)
		 + x4d(n, 0, h_id + 4, w_id + 6) * k4d(m, 0, 4, 6)
											  
		 + x4d(n, 0, h_id + 5, w_id + 0) * k4d(m, 0, 5, 0)
		 + x4d(n, 0, h_id + 5, w_id + 1) * k4d(m, 0, 5, 1)
		 + x4d(n, 0, h_id + 5, w_id + 2) * k4d(m, 0, 5, 2)
		 + x4d(n, 0, h_id + 5, w_id + 3) * k4d(m, 0, 5, 3)
		 + x4d(n, 0, h_id + 5, w_id + 4) * k4d(m, 0, 5, 4)
		 + x4d(n, 0, h_id + 5, w_id + 5) * k4d(m, 0, 5, 5)
		 + x4d(n, 0, h_id + 5, w_id + 6) * k4d(m, 0, 5, 6)
											  
		 + x4d(n, 0, h_id + 6, w_id + 0) * k4d(m, 0, 6, 0)
		 + x4d(n, 0, h_id + 6, w_id + 1) * k4d(m, 0, 6, 1)
		 + x4d(n, 0, h_id + 6, w_id + 2) * k4d(m, 0, 6, 2)
		 + x4d(n, 0, h_id + 6, w_id + 3) * k4d(m, 0, 6, 3)
		 + x4d(n, 0, h_id + 6, w_id + 4) * k4d(m, 0, 6, 4)
		 + x4d(n, 0, h_id + 6, w_id + 5) * k4d(m, 0, 6, 5)
		 + x4d(n, 0, h_id + 6, w_id + 6) * k4d(m, 0, 6, 6);
	}
#undef y4d
#undef x4d
#undef k4d
}
	
__host__ void GPUInterface::conv_forward_gpu_prolog(const float *__restrict host_y, const float *__restrict host_x, const float *__restrict host_k, float **__restrict device_y_ptr, float **__restrict device_x_ptr, float **__restrict device_k_ptr, const int B, const int M, const int C, const int H, const int W, const int K)
{
    // Allocate memory and copy over the relevant data structures to the GPU

    // We pass double pointers for you to initialize the relevant device pointers,
    //  which are passed to the other two functions.

    // Useful snippet for error checking
	
	int size_k = M * C * K * K * sizeof(float);
	
	hipMalloc((void **) device_y_ptr, B * M * (H - K + 1) * (W - K + 1) * sizeof(float));
	hipMalloc((void **) device_x_ptr, B * C * H * W * sizeof(float));
	//hipMalloc((void **) device_k_ptr, size_k);
	
	for(int i = 0; i < nStreams; ++i)
		hipStreamCreate(&stream[i]);
	
	const int streamSize1 = B*M*(H-K+1)*(W-K+1)/nStreams;
	const int streamSize2 = B*C*H*W/nStreams;
	
	for (int i = 0; i < nStreams; ++i)
	{
		int offset1 = i * streamSize1;
		int offset2 = i * streamSize2;
		hipMemcpyAsync(*device_y_ptr+offset1, (void *)(host_y+offset1), B*M*(H-K+1)*(W-K+1)*sizeof(float)/nStreams, hipMemcpyHostToDevice, stream[i]);
		hipMemcpyAsync(*device_x_ptr+offset2, (void *)(host_x+offset2), B*C*H*W*sizeof(float)/nStreams, hipMemcpyHostToDevice, stream[i]);
	}
	//hipMemcpy(*device_k_ptr, (void *)host_k, size_k, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(deviceKernel), host_k, size_k, 0, hipMemcpyHostToDevice);
	//hipError_t error = hipGetLastError();
    //if(error != hipSuccess)
    //{
    //    std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
    //    exit(-1);
    //}
}


__host__ void GPUInterface::conv_forward_gpu(float *__restrict device_y, const float *__restrict device_x, const float *__restrict device_k, const int B, const int M, const int C, const int H, const int W, const int K)
{
    // Set the kernel dimensions and call the kernel
	const int H_out = H - K + 1;
    const int W_out = W - K + 1;
	const int H_grid = ceil(H_out/float(BLOCK_WIDTH));
	const int W_grid = ceil(W_out/float(BLOCK_WIDTH));
	
	const int streamSize = B/nStreams;
	dim3 dimGrid(B/nStreams, M, H_grid*W_grid);
	dim3 dimBlock(BLOCK_WIDTH, BLOCK_WIDTH, 1);
	for (int i = 0; i < nStreams; ++i)
	{
		int offset = i * streamSize;
		if(C == 1)
			conv_forward_kernel_1<<<dimGrid, dimBlock, 0, stream[i]>>>(device_y, device_x, B, M, C, H, W, K, offset);
		else
			conv_forward_kernel_4<<<dimGrid, dimBlock, 0, stream[i]>>>(device_y, device_x, B, M, C, H, W, K, offset);
	}
	hipDeviceSynchronize();
	//hipError_t error = hipGetLastError();
    //if(error != hipSuccess)
    //{
    //    std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
    //    exit(-1);
    //}
}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *__restrict host_y, float *__restrict device_y, float *__restrict device_x, float *__restrict device_k, const int B, const int M, const int C, const int H, const int W, const int K)
{
    // Copy the output back to host
	const int streamSize = B*M*(H - K + 1)*(W - K + 1)/nStreams;
	for (int i = 0; i < nStreams; ++i)
	{
		int offset = i * streamSize;
		hipMemcpyAsync(host_y+offset, device_y+offset, B*M*(H - K + 1)*(W - K + 1)*sizeof(float)/nStreams, hipMemcpyDeviceToHost, stream[i]);
	}
    // Free device memory
	hipFree(device_y);
	hipFree(device_x);
	//hipFree(device_k);
	//hipError_t error = hipGetLastError();
    //if(error != hipSuccess)
    //{
    //    std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
    //    exit(-1);
    //}
}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}
